
#include <hip/hip_runtime.h>
///
/// vecAddKernel00.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
/// 

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int tsize   = N;
    int bsize   = tsize * blockDim.x;
    int bid     = bsize * blockIdx.x;
    int tid     = threadIdx.x;
    
    int startAdr= bid + tid;
    int endAdr  = bid + bsize;
    int gap     = blockDim.x;
    int cur;
    for ( cur = startAdr; cur < endAdr; cur += gap )
        C[cur] = A[cur] + B[cur];
}

// blockIdx.x: i-th block in a grid that stores n threads
// blockDim.x: size of a block (ie. # threads per block)
// threadIdx.x: j-th element of a thread.
// gridDim.x: size of a grid (ie. # blocks per grid)
// threadDim.x: N values per thread

// 60/128/60000

// all threads in each block starts from 0~(blockDim.x-1)
// bid = loc * 1 block size
// 1 block size = # threads * 1 thread size
