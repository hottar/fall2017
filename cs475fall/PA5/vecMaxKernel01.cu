///
/// vecMaxKernel00.cu : code computes partiol answer per each thread. 
/// By Waruna Ranasinghe 
/// Created: 15 Aug 2017
/// Last Modified: 

/// The code computes partial answer for reduction over max per a thread. 
/// Each thread computes max of a consective chunck of data of size C.
/// The memory loads are not coalesced. 


#include <hip/hip_runtime.h>
#include <stdio.h>
/*
 * A - input vector of floats of size G*B*C
 * reductions - output of partial answers compted by each thread
 * C - chunck size - number of elements processed by ech thread
 */
__global__ void reduce(const float* A, float* reductions, int C) {
    int tsize   = 125000; // units per thread
    int bsize   = tsize * blockDim.x; // units per block
    int bid     = bsize * blockIdx.x; // pos of cur block
    int tid     = threadIdx.x; // initial pos of thread
    int startAdr= tid + bid;
    int endAdr  = bid + bsize;
    int reducAt = blockIdx.x * blockDim.x + threadIdx.x;
    int gap     = blockDim.x;
    int k;
    for ( k = startAdr; k < endAdr; k+=gap )
        reductions[reducAt] = max(reductions[reducAt],A[k]);
}
